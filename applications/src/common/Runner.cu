#include "hip/hip_runtime.h"
#include "Runner.hpp"

#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include <algorithm>
#include <numeric>

SortMethod sortToValidateMethod;

template <class T>
__global__ void thrustSortKernel(T* dChromosome,
                                 unsigned* dPermutation,
                                 unsigned length) {
  thrust::device_ptr<T> keys(dChromosome);
  thrust::device_ptr<unsigned> vals(dPermutation);
  thrust::sort_by_key(thrust::device, keys, keys + length, vals);
}

template <class T>
void sortChromosomeToValidateImpl(const T* chromosome,
                                  unsigned* permutation,
                                  unsigned length) {
  auto method = sortToValidateMethod;
  std::iota(permutation, permutation + length, 0);

  if (method == SortMethod::stdSort) {
    std::sort(permutation, permutation + length, [&](unsigned a, unsigned b) {
      return chromosome[a] < chromosome[b];
    });
    return;
  }

  // gpu sorting

  T* dChromosome = nullptr;
  hipMalloc(&dChromosome, length * sizeof(T));
  hipMemcpy(dChromosome, chromosome, length * sizeof(T),
             hipMemcpyHostToDevice);

  unsigned* dPermutation = nullptr;
  hipMalloc(&dPermutation, length * sizeof(unsigned));
  hipMemcpy(dPermutation, permutation, length * sizeof(unsigned),
             hipMemcpyHostToDevice);

  if (method == SortMethod::bbSegSort) {
    assert(sizeof(T) == sizeof(float));
    bbSegSortCall((float*)dChromosome, dPermutation, length);
  } else if (method == SortMethod::thrustHost) {
    thrust::device_ptr<T> keys(dChromosome);
    thrust::device_ptr<unsigned> vals(dPermutation);
    thrust::sort_by_key(thrust::device, keys, keys + length, vals);
  } else if (method == SortMethod::thrustKernel) {
    thrustSortKernel<<<1, 1>>>(dChromosome, dPermutation, length);
  } else {
    std::cerr << __PRETTY_FUNCTION__ << ": not implemented for method "
              << method << std::endl;
    abort();
  }
  hipDeviceSynchronize();

  hipMemcpy(permutation, dPermutation, length * sizeof(unsigned),
             hipMemcpyDeviceToHost);

  hipFree(dChromosome);
  hipFree(dPermutation);
}

void sortChromosomeToValidate(const float* chromosome,
                              unsigned* permutation,
                              unsigned length) {
  sortChromosomeToValidateImpl(chromosome, permutation, length);
}

void sortChromosomeToValidate(const double* chromosome,
                              unsigned* permutation,
                              unsigned length) {
  sortChromosomeToValidateImpl(chromosome, permutation, length);
}
