#include "hip/hip_runtime.h"
#include "TspInstance.hpp"
#include <brkga_cuda_api/BBSegSort.cuh>
#include <brkga_cuda_api/CudaUtils.hpp>

#include <hip/hip_runtime.h>

#include <algorithm>
#include <cstdio>
#include <iostream>
#include <numeric>
#include <vector>

float hostDecode(const float* chromosome,
                 const unsigned n,
                 const std::vector<float>& distances) {
  std::vector<unsigned> indices(n);
  std::iota(indices.begin(), indices.end(), 0);
  std::sort(indices.begin(), indices.end(),
            [&](int a, int b) { return chromosome[a] < chromosome[b]; });

  float fitness = distances[indices[0] * n + indices[n - 1]];
  for (unsigned i = 1; i < n; ++i)
    fitness += distances[indices[i - 1] * n + indices[i]];
  return fitness;
}

void TspInstance::evaluateChromosomesOnHost(const unsigned numberOfChromosomes,
                                            const float* chromosomes,
                                            float* results) const {
  for (unsigned i = 0; i < numberOfChromosomes; ++i)
    results[i] = hostDecode(chromosomes + i * chromosomeLength(),
                            chromosomeLength(), distances);
}

void TspInstance::evaluateChromosomesOnDevice(hipStream_t stream,
                                              unsigned numberOfChromosomes,
                                              const float* dChromosomes,
                                              float* dResults) const {
  const auto length = numberOfChromosomes * chromosomeLength();
  auto* keys = cuda::alloc<float>(length);
  auto* indices = cuda::alloc<unsigned>(length);

  cuda::copy(stream, keys, dChromosomes, length);
  cuda::iotaMod(stream, indices, length, chromosomeLength(), threadsPerBlock);
  cuda::segSort(keys, indices, length, chromosomeLength());

  evaluateIndicesOnDevice(stream, numberOfChromosomes, indices, dResults);

  cuda::free(keys);
  cuda::free(indices);
}

__device__ float deviceDecodeSorted(const unsigned* indices,
                                    const unsigned n,
                                    const float* distances) {
  float fitness = distances[indices[0] * n + indices[n - 1]];
  for (unsigned i = 1; i < n; ++i)
    fitness += distances[indices[i - 1] * n + indices[i]];
  return fitness;
}

__global__ void tspDecodeSorted(const unsigned numberOfChromosomes,
                                const unsigned chromosomeLength,
                                const float* distances,
                                const unsigned* indices,
                                float* results) {
  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= numberOfChromosomes) return;

  auto* curIndices = indices + tid * chromosomeLength;
  results[tid] = deviceDecodeSorted(curIndices, chromosomeLength, distances);
}

void TspInstance::evaluateIndicesOnDevice(hipStream_t stream,
                                          const unsigned numberOfChromosomes,
                                          const unsigned* dIndices,
                                          float* dResults) const {
  const auto threads = threadsPerBlock;
  const auto blocks = (numberOfChromosomes + threads - 1) / threads;
  tspDecodeSorted<<<blocks, threads, 0, stream>>>(
      numberOfChromosomes, chromosomeLength(), dDistances, dIndices, dResults);
}
