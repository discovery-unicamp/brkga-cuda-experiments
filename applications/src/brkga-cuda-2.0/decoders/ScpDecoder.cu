#include "hip/hip_runtime.h"
#include "../../common/instances/ScpInstance.hpp"
#include "ScpDecoder.hpp"
#include <brkga-cuda/Chromosome.hpp>
#include <brkga-cuda/CudaUtils.hpp>

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

#include <cmath>
#include <limits>
#include <vector>

ScpDecoder::ScpDecoder(ScpInstance* _instance)
    : instance(_instance),
      dCosts(box::cuda::alloc<float>(nullptr, instance->costs.size())),
      dSets(nullptr),
      dSetEnd(nullptr) {
  box::cuda::copy2d(nullptr, dCosts, instance->costs.data(),
                    instance->costs.size());

  const auto& sets = instance->sets;
  dSets = box::cuda::alloc<unsigned>(nullptr, sets.size());
  box::cuda::copy2d(nullptr, dSets, sets.data(), sets.size());

  const auto& setsEnd = instance->setsEnd;
  dSetEnd = box::cuda::alloc<unsigned>(nullptr, setsEnd.size());
  box::cuda::copy2d(nullptr, dSetEnd, setsEnd.data(), setsEnd.size());
}

ScpDecoder::~ScpDecoder() {
  box::cuda::free(nullptr, dCosts);
  box::cuda::free(nullptr, dSets);
  box::cuda::free(nullptr, dSetEnd);
}

float ScpDecoder::decode(const box::Chromosome<float>& chromosome) const {
  return getFitness(chromosome, config->chromosomeLength, instance->universeSize,
              instance->acceptThreshold, instance->costs.data(),
              instance->sets.data(), instance->setsEnd.data());
}

__global__ void deviceDecode(float* results,
                             const unsigned numberOfChromosomes,
                             const box::Chromosome<float>* dChromosomes,
                             const unsigned n,
                             const unsigned universeSize,
                             const float threshold,
                             const float* dCosts,
                             const unsigned* dSets,
                             const unsigned* dSetsEnd) {
  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= numberOfChromosomes) return;
  results[tid] = getFitness(dChromosomes[tid], n, universeSize, threshold,
                            dCosts, dSets, dSetsEnd);
}

void ScpDecoder::decode(hipStream_t stream,
                        unsigned numberOfChromosomes,
                        const box::Chromosome<float>* dChromosomes,
                        float* dResults) const {
  const auto threads = config->threadsPerBlock;
  const auto blocks = box::cuda::blocks(numberOfChromosomes, threads);
  deviceDecode<<<blocks, threads, 0, stream>>>(
      dResults, numberOfChromosomes, dChromosomes, config->chromosomeLength,
      instance->universeSize, instance->acceptThreshold, dCosts, dSets,
      dSetEnd);
  CUDA_CHECK_LAST();
}
