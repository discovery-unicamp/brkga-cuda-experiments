#include "hip/hip_runtime.h"
#include "../../common/instances/ScpInstance.cuh"
#include "ScpDecoder.hpp"
#include <brkga-cuda/CudaUtils.hpp>

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

#include <cmath>
#include <limits>
#include <vector>

ScpDecoder::ScpDecoder(ScpInstance* _instance)
    : instance(_instance),
      dCosts(box::cuda::alloc<float>(nullptr, instance->costs.size())),
      dSets(nullptr),
      dSetEnd(nullptr) {
  box::cuda::copy2d(nullptr, dCosts, instance->costs.data(),
                    instance->costs.size());

  std::vector<unsigned> tempSets;
  std::vector<unsigned> tempSetEnd;
  for (auto set : instance->sets) {
    tempSetEnd.push_back(tempSetEnd.empty() ? 0 : tempSetEnd.back());
    for (auto element : set) {
      tempSets.push_back(element);
      ++tempSetEnd.back();
    }
  }

  dSets = box::cuda::alloc<unsigned>(nullptr, tempSets.size());
  box::cuda::copy2d(nullptr, dSets, tempSets.data(), tempSets.size());

  dSetEnd = box::cuda::alloc<unsigned>(nullptr, tempSetEnd.size());
  box::cuda::copy2d(nullptr, dSetEnd, tempSetEnd.data(), tempSetEnd.size());
}

ScpDecoder::~ScpDecoder() {
  box::cuda::free(nullptr, dCosts);
  box::cuda::free(nullptr, dSets);
  box::cuda::free(nullptr, dSetEnd);
}

float ScpDecoder::decode(const box::Chromosome<float>& chromosome) const {
  const auto n = config->chromosomeLength;
  const auto& costs = instance->costs;
  const auto& sets = instance->sets;

  float fitness = 0;
  std::vector<bool> covered(instance->universeSize);
  unsigned numCovered = 0;
  for (unsigned i = 0; i < n; ++i) {
    if (chromosome[i] > ScpInstance::ACCEPT_THRESHOLD) {
      fitness += costs[i];
      for (auto element : sets[i]) {
        if (!covered[element]) {
          covered[element] = true;
          ++numCovered;
        }
      }
    }
  }

  if (numCovered != instance->universeSize)
    return std::numeric_limits<float>::infinity();
  return fitness;
}

__global__ void deviceDecode(float* results,
                             const unsigned numberOfChromosomes,
                             const box::Chromosome<float>* dChromosomes,
                             const unsigned n,
                             const unsigned universeSize,
                             const float threshold,
                             const float* dCosts,
                             const unsigned* dSets,
                             const unsigned* dSetEnd) {
  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= numberOfChromosomes) return;

  const auto& chromosome = dChromosomes[tid];

  unsigned numCovered = 0;
  bool* covered = new bool[universeSize];
  for (unsigned i = 0; i < universeSize; ++i) covered[i] = false;

  float fitness = 0;
  for (unsigned i = 0; i < n; ++i) {
    if (chromosome[i] > threshold) {
      fitness += dCosts[i];
      for (unsigned j = (i == 0 ? 0 : dSetEnd[i - 1]); j < dSetEnd[i]; ++j) {
        if (!covered[dSets[j]]) {
          covered[dSets[j]] = true;
          ++numCovered;
        }
      }
    }
  }

  delete[] covered;
  results[tid] = numCovered != universeSize ? INFINITY : fitness;
}

void ScpDecoder::decode(hipStream_t stream,
                        unsigned numberOfChromosomes,
                        const box::Chromosome<float>* dChromosomes,
                        float* dResults) const {
  const auto threads = config->threadsPerBlock;
  const auto blocks = box::cuda::blocks(numberOfChromosomes, threads);
  deviceDecode<<<blocks, threads, 0, stream>>>(
      dResults, numberOfChromosomes, dChromosomes, config->chromosomeLength,
      instance->universeSize, ScpInstance::ACCEPT_THRESHOLD, dCosts, dSets,
      dSetEnd);
  CUDA_CHECK_LAST();
}
