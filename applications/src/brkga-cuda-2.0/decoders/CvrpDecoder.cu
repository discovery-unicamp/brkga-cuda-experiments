#include "hip/hip_runtime.h"
#include "../../common/instances/CvrpInstance.hpp"
#include "CvrpDecoder.hpp"
#include <brkga-cuda/Chromosome.hpp>
#include <brkga-cuda/utils/GpuUtils.hpp>

#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include <algorithm>
#include <numeric>
#include <vector>

CvrpDecoder::CvrpDecoder(CvrpInstance* _instance)
    : instance(_instance),
      dDemands(box::gpu::alloc<unsigned>(nullptr, instance->demands.size())),
      dDistances(box::gpu::alloc<float>(nullptr, instance->distances.size())) {
  box::gpu::copy2d(nullptr, dDemands, instance->demands.data(),
                   instance->demands.size());
  box::gpu::copy2d(nullptr, dDistances, instance->distances.data(),
                   instance->distances.size());
}

CvrpDecoder::~CvrpDecoder() {
  box::gpu::free(nullptr, dDemands);
  box::gpu::free(nullptr, dDistances);
}

box::Fitness CvrpDecoder::decode(
    const box::Chromosome<box::Gene>& chromosome) const {
  std::vector<unsigned> permutation(config->chromosomeLength());
  std::iota(permutation.begin(), permutation.end(), 0);
  std::sort(permutation.begin(), permutation.end(),
            [&chromosome](unsigned a, unsigned b) {
              return chromosome[a] < chromosome[b];
            });
  return getFitness(permutation.data(), config->chromosomeLength(),
                    instance->capacity, instance->demands.data(),
                    instance->distances.data());
}

box::Fitness CvrpDecoder::decode(
    const box::Chromosome<box::GeneIndex>& permutation) const {
  return getFitness(permutation, config->chromosomeLength(), instance->capacity,
                    instance->demands.data(), instance->distances.data());
}

__global__ void deviceDecode(box::Fitness* dFitness,
                             box::uint numberOfChromosomes,
                             box::Gene* dChromosomes,
                             unsigned* dTempMemory,
                             unsigned chromosomeLength,
                             unsigned capacity,
                             const unsigned* dDemands,
                             const float* dDistances) {
  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= numberOfChromosomes) return;

  auto* chromosome = dChromosomes + tid * chromosomeLength;
  auto* tour = dTempMemory + tid * chromosomeLength;
  for (unsigned i = 0; i < chromosomeLength; ++i) tour[i] = i;

  thrust::device_ptr<box::Gene> keys(chromosome);
  thrust::device_ptr<unsigned> vals(tour);
  thrust::sort_by_key(thrust::device, keys, keys + chromosomeLength, vals);

  dFitness[tid] =
      getFitness(tour, chromosomeLength, capacity, dDemands, dDistances);
}

void CvrpDecoder::decode(hipStream_t stream,
                         box::uint numberOfChromosomes,
                         const box::Chromosome<box::Gene>* dChromosomes,
                         box::Fitness* dFitness) const {
  const auto length = numberOfChromosomes * config->chromosomeLength();
  auto* dChromosomesCopy = box::gpu::alloc<box::Gene>(stream, length);
  auto* dTempMemory = box::gpu::alloc<unsigned>(stream, length);

  box::Chromosome<box::Gene>::copy(stream, dChromosomesCopy, dChromosomes,
                                   numberOfChromosomes,
                                   config->chromosomeLength());

  const auto threads = config->gpuThreads();
  const auto blocks = box::gpu::blocks(numberOfChromosomes, threads);
  deviceDecode<<<blocks, threads, 0, stream>>>(
      dFitness, numberOfChromosomes, dChromosomesCopy, dTempMemory,
      config->chromosomeLength(), instance->capacity, dDemands, dDistances);
  CUDA_CHECK_LAST();

  box::gpu::free(stream, dChromosomesCopy);
  box::gpu::free(stream, dTempMemory);
}

__global__ void deviceDecode(box::Fitness* dFitness,
                             box::uint tourCount,
                             const box::Chromosome<box::GeneIndex>* tourList,
                             unsigned chromosomeLength,
                             unsigned capacity,
                             const unsigned* dDemands,
                             const float* dDistances) {
  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= tourCount) return;

  const auto& tour = tourList[tid];
  dFitness[tid] =
      getFitness(tour, chromosomeLength, capacity, dDemands, dDistances);
}

void CvrpDecoder::decode(hipStream_t stream,
                         box::uint numberOfPermutations,
                         const box::Chromosome<box::GeneIndex>* dPermutations,
                         box::Fitness* dFitness) const {
  const auto threads = config->gpuThreads();
  const auto blocks = box::gpu::blocks(numberOfPermutations, threads);
  deviceDecode<<<blocks, threads, 0, stream>>>(
      dFitness, numberOfPermutations, dPermutations, config->chromosomeLength(),
      instance->capacity, dDemands, dDistances);
  CUDA_CHECK_LAST();
}
