#include "hip/hip_runtime.h"
#include "../Tweaks.hpp"  // Must be generated
#include "../common/Checker.hpp"
#include "../common/Parameters.hpp"
#include "CudaCheck.cuh"
#include <GPU-BRKGA/GPUBRKGA.cuh>

#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include <iomanip>
#include <iostream>
#include <vector>

#if defined(TSP)
#include "../common/instances/TspInstance.hpp"
#include "decoders/TspDecoder.hpp"
typedef TspInstance Instance;
typedef TspDecoder DecoderImpl;
#elif defined(SCP)
#include "../common/instances/ScpInstance.hpp"
#include "decoders/ScpDecoder.hpp"
typedef ScpInstance Instance;
typedef ScpDecoder DecoderImpl;
#elif defined(CVRP) || defined(CVRP_GREEDY)
#include "../common/instances/CvrpInstance.hpp"
#include "decoders/CvrpDecoder.hpp"
typedef CvrpInstance Instance;
typedef CvrpDecoder DecoderImpl;
#else
#error No problem/instance/decoder defined
#endif  // Problem/Instance

std::string decodeType;

inline bool contains(const std::string& str, const std::string& pattern) {
  return str.find(pattern) != std::string::npos;
}

__global__ void callSort(float* dChromosome,
                         unsigned* dPermutation,
                         unsigned chromosomeLength) {
  thrust::device_ptr<float> keys(dChromosome);
  thrust::device_ptr<unsigned> vals(dPermutation);
  thrust::sort_by_key(thrust::device, keys, keys + chromosomeLength, vals);
}

void sortChromosomeToValidate(const float* chromosome,
                              unsigned* permutation,
                              unsigned size) {
  std::iota(permutation, permutation + size, 0);

  if (contains(decodeType, "gpu")) {
    // Uses thrust::sort
    float* dChromosome = nullptr;
    unsigned* dPermutation = nullptr;

    CUDA_CHECK(hipMalloc(&dChromosome, size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&dPermutation, size * sizeof(unsigned)));

    CUDA_CHECK(hipMemcpy(dChromosome, chromosome, size * sizeof(float),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dPermutation, permutation, size * sizeof(unsigned),
                          hipMemcpyHostToDevice));

    callSort<<<1, 1>>>(dChromosome, dPermutation, size);
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(permutation, dPermutation, size * sizeof(unsigned),
                          hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(dChromosome));
    CUDA_CHECK(hipFree(dPermutation));
  } else if (contains(decodeType, "cpu")) {
    // Uses std::sort
    std::sort(permutation, permutation + size, [&](unsigned a, unsigned b) {
      return chromosome[a] < chromosome[b];
    });
  } else {
    std::cerr << __PRETTY_FUNCTION__ << ": unknown decoder `" << decodeType
              << "`\n";
    abort();
  }
}

void sortChromosomeToValidate(const double*, unsigned*, unsigned) {
  std::cerr << __PRETTY_FUNCTION__ << " should not be called\n";
  abort();
}

template <class T>
float getBestFitness(GPUBRKGA<T>& brkga) {
  auto best = brkga.getBestIndividual();
  CUDA_CHECK_LAST();
  return best.fitness.first;
}

template <class T>
std::pair<float, std::vector<float>> getBest(GPUBRKGA<T>& brkga,
                                             unsigned length) {
  auto best = brkga.getBestIndividual();
  CUDA_CHECK_LAST();
  auto fitness = best.fitness.first;
  auto chromosome = std::vector<float>(best.aleles, best.aleles + length);
  return {fitness, chromosome};
}

int main(int argc, char** argv) {
  auto params = Parameters::parse(argc, argv);
  decodeType = params.decoder;

  Instance instance = Instance::fromFile(params.instanceFileName);
  DecoderImpl decoder(&instance, params);

  CHECK(params.decoder == "cpu" || params.decoder == "gpu",
        "Unsupported decoder: %s", params.decoder.c_str());

  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  CUDA_CHECK(hipEventRecord(start));

  GPUBRKGA<DecoderImpl> brkga(
      instance.chromosomeLength(), params.populationSize,
      params.getEliteProportion(), params.getMutantProportion(), params.rhoe,
      decoder, params.seed, /* decode on gpu? */ params.decoder == "gpu",
      params.numberOfPopulations);

  std::vector<float> convergence;
  convergence.push_back(getBestFitness(brkga));
  for (unsigned gen = 1; gen <= params.generations; ++gen) {
    brkga.evolve();
    if (gen % params.exchangeBestInterval == 0 && gen != params.generations)
      brkga.exchangeElite(params.exchangeBestCount);
    if (gen % params.logStep == 0 || gen == params.generations) {
      float best = getBestFitness(brkga);
      std::clog << "Generation " << gen << "; best: " << best << "        \r";
      convergence.push_back(best);
    }
  }
  std::clog << '\n';

  float bestFitness = -1;
  std::vector<float> bestChromosome;
  std::tie(bestFitness, bestChromosome) =
      getBest(brkga, instance.chromosomeLength());

  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));
  float timeElapsedMs = -1;
  hipEventElapsedTime(&timeElapsedMs, start, stop);

  instance.validate(bestChromosome.data(), bestFitness);

  std::cout << std::fixed << std::setprecision(6) << "ans=" << bestFitness
            << " elapsed=" << timeElapsedMs / 1000 << " convergence=";
  bool flag = 0;
  std::cout << "[";
  for (auto x : convergence) {
    if (flag) std::cout << ",";
    flag = true;
    std::cout << x;
  }
  std::cout << "]\n";

  return 0;
}
