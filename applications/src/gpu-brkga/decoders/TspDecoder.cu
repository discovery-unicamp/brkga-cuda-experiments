#include "hip/hip_runtime.h"
#include "../../common/instances/TspInstance.hpp"
#include "TspDecoder.hpp"
#include <brkga-cuda/utils/GpuUtils.hpp>

#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include <algorithm>
#include <numeric>
#include <vector>

TspDecoder::TspDecoder(TspInstance* _instance, const Parameters& params)
    : GpuBrkga::Decoder(
        params.populationSize,
        _instance->chromosomeLength(),
        (params.decoder == "cpu" ? params.ompThreads : params.threadsPerBlock),
        params.decoder == "cpu"),
      instance(_instance),
      dDistances(nullptr) {
  if (!isCpuDecode) {
    CUDA_CHECK(
        hipMalloc(&dDistances, instance->distances.size() * sizeof(float)));
    CUDA_CHECK(hipMemcpy(dDistances, instance->distances.data(),
                          instance->distances.size() * sizeof(float),
                          hipMemcpyHostToDevice));

    // Set CUDA heap limit to 1GB to avoid memory issues with the sort of thrust
    CUDA_CHECK(hipDeviceSetLimit(hipLimitMallocHeapSize,
                                  (std::size_t)1024 * 1024 * 1024));
  }
}

TspDecoder::~TspDecoder() {
  if (!isCpuDecode) CUDA_CHECK(hipFree(dDistances));
}

TspDecoder::Fitness TspDecoder::DecodeOnCpu(const float* chromosome) const {
  std::vector<unsigned> permutation(chromosomeLength);
  std::iota(permutation.begin(), permutation.end(), 0);
  std::sort(permutation.begin(), permutation.end(),
            [chromosome](unsigned a, unsigned b) {
              return chromosome[a] < chromosome[b];
            });
  return getFitness(permutation.data(), chromosomeLength,
                    instance->distances.data());
}

__global__ void deviceDecodeKernel(const unsigned numberOfChromosomes,
                                   float* dChromosomes,
                                   unsigned* dTempMemory,
                                   const unsigned chromosomeLength,
                                   const float* dDistances,
                                   float* dFitness) {
  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= numberOfChromosomes) return;

  float* chromosome = dChromosomes + tid * chromosomeLength;
  unsigned* tour = dTempMemory + tid * chromosomeLength;
  for (unsigned i = 0; i < chromosomeLength; ++i) tour[i] = i;

  thrust::device_ptr<float> keys(chromosome);
  thrust::device_ptr<unsigned> vals(tour);
  thrust::sort_by_key(thrust::device, keys, keys + chromosomeLength, vals);

  dFitness[tid] = getFitness(tour, chromosomeLength, dDistances);
}

void TspDecoder::DecodeOnGpu(const float* dChromosomes, float* dFitness) const {
  assert(!isCpuDecode);
  const auto length = populationSize * chromosomeLength;

  float* dChromosomesCopy = nullptr;
  CUDA_CHECK(hipMalloc(&dChromosomesCopy, length * sizeof(float)));
  CUDA_CHECK(hipMemcpy(dChromosomesCopy, dChromosomes, length * sizeof(float),
                        hipMemcpyDeviceToDevice));

  unsigned* dTempMemory = nullptr;
  CUDA_CHECK(hipMalloc(&dTempMemory, length * sizeof(unsigned)));

  const auto threads = numberOfThreads;
  const auto blocks = (populationSize + threads - 1) / threads;
  deviceDecodeKernel<<<blocks, threads>>>(populationSize, dChromosomesCopy,
                                          dTempMemory, chromosomeLength,
                                          dDistances, dFitness);
  CUDA_CHECK_LAST();

  CUDA_CHECK(hipFree(dChromosomesCopy));
  CUDA_CHECK(hipFree(dTempMemory));
}
