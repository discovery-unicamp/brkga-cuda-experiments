#include "hip/hip_runtime.h"
// Copyright (c) 2021 Bruno Almêda de Oliveira. All rights reserved.
// Released under the GNU General Public License version 2 or later.

#include "CvrpInstance.hpp"

CvrpInstance::Solution::Solution(const CvrpInstance& instance, float newFitness, std::vector<unsigned> newTour)
    : fitness(newFitness), tour(std::move(newTour)) {
  if (tour.empty())
    throw std::runtime_error("Tour is empty");
  if (tour[0] != 0)
    throw std::runtime_error("Tour should start at depot (0)");
  if (tour.back() != 0)
    throw std::runtime_error("Tour should finish at depot (0)");

  std::vector<bool> visited(instance.numberOfClients + 1);
  for (unsigned u : tour) {
    if (u > instance.numberOfClients)
      throw std::runtime_error("Invalid client in the tour");
    if (u != 0 && visited[u])
      throw std::runtime_error("Client was visited twice");
    visited[u] = true;
  }
  if (!std::all_of(visited.begin(), visited.end(), [](bool x) { return x; })) {
    throw std::runtime_error("Missing clients in the tour");
  }

  for (unsigned i = 1; i < tour.size(); ++i) {
    if (tour[i - 1] == tour[i])
      throw std::runtime_error("Found an empty tour");
  }

  unsigned filled = 0;
  for (unsigned u : tour) {
    if (u == 0) {
      filled = 0;
    } else {
      filled += instance.demands[u];
      if (filled > instance.capacity)
        throw std::runtime_error("Capacity exceeded");
    }
  }

  float expectedFitness = 0;
  for (unsigned i = 1; i < tour.size(); ++i) {
    unsigned u = tour[i - 1];
    unsigned v = tour[i];
    expectedFitness += instance.distances[u * (instance.numberOfClients + 1) + v];
  }
  if (std::abs(fitness - expectedFitness) > 1e-3)
    throw std::runtime_error("Invalid fitness");
}

CvrpInstance CvrpInstance::fromFile(const std::string& filename) {
  std::ifstream file(filename);
  if (!file.is_open())
    throw std::runtime_error("Failed to open file " + filename);

  CvrpInstance instance;
  std::string str;

  // read capacity
  while ((file >> str) && str != "NODE_COORD_SECTION") {
    if (str == "CAPACITY") {
      file >> str >> instance.capacity;
    } else if (str == "DIMENSION") {
      file >> str >> instance.numberOfClients;
      --instance.numberOfClients;
    } else if (str == "NAME") {
      file >> str >> instance.name;
    }
  }

  // read locations
  while ((file >> str) && str != "DEMAND_SECTION") {
    float x, y;
    file >> x >> y;
    instance.locations.push_back({x, y});
  }
  instance.numberOfClients = instance.locations.size() - 1;

  // read demands
  while ((file >> str) && str != "DEPOT_SECTION") {
    int d;
    file >> d;
    instance.demands.push_back(d);
  }
  const auto demandsSize = instance.demands.size() * sizeof(int);
  CUDA_CHECK(hipMalloc(&instance.dDemands, demandsSize));
  CUDA_CHECK(hipMemcpy(instance.dDemands, instance.demands.data(), demandsSize, hipMemcpyHostToDevice));

  assert(!instance.name.empty());
  assert(instance.numberOfClients != static_cast<unsigned>(-1));      // no dimension
  assert(instance.capacity != static_cast<unsigned>(-1));             // no capacity
  assert(instance.locations.size() > 1);                              // no client provided
  assert(instance.locations.size() == instance.numberOfClients + 1);  // missing location
  assert(instance.demands.size() == instance.numberOfClients + 1);    // missing demand
  assert(instance.demands[0] == 0);                                   // depot has demand
  assert(std::all_of(instance.demands.begin() + 1, instance.demands.end(),
                     [](int d) { return d > 0; }));  // client wo/ demand

  const auto n = instance.numberOfClients;
  instance.distances.resize((n + 1) * (n + 1));
  for (unsigned i = 0; i <= n; ++i)
    for (unsigned j = 0; j <= n; ++j)
      instance.distances[i * (n + 1) + j] = instance.locations[i].distance(instance.locations[j]);

  const auto distancesSize = instance.distances.size() * sizeof(float);
  CUDA_CHECK(hipMalloc(&instance.dDistances, distancesSize));
  CUDA_CHECK(hipMemcpy(instance.dDistances, instance.distances.data(), distancesSize, hipMemcpyHostToDevice));

  return instance;
}

CvrpInstance::~CvrpInstance() {
  CUDA_CHECK(hipFree(dDistances));
  CUDA_CHECK(hipFree(dDemands));
}

void CvrpInstance::validateBestKnownSolution(const std::string& filename) {
  std::cerr << "Reading best known solution from " << filename << '\n';
  std::ifstream file(filename);
  assert(file.is_open());
  std::string line;

  std::vector<unsigned> tour;
  tour.push_back(0);  // start on the depot
  while (std::getline(file, line) && line.rfind("Route") == 0) {
    std::stringstream ss(line);

    std::string tmp;
    ss >> tmp >> tmp;

    unsigned u;
    while (ss >> u)
      tour.push_back(u);
    tour.push_back(0);  // return to the depot
  }

  assert(line.rfind("Cost") == 0);
  float fitness = std::stof(line.substr(5));

  Solution(*this, fitness, tour);
}

CvrpInstance::Solution CvrpInstance::convertChromosomeToSolution(const float* chromosome) const {
  const auto clen = chromosomeLength();
  std::vector<unsigned> indices(clen);
  std::iota(indices.begin(), indices.end(), 0);
  std::sort(indices.begin(), indices.end(), [&](int a, int b) { return chromosome[a] < chromosome[b]; });

  unsigned filled = 0;
  std::vector<unsigned> tour;
  tour.push_back(0);  // start in the depot
  for (unsigned k = 0; k < clen; ++k) {
    unsigned v = indices[k] + 1;
    if (filled + demands[v] > capacity) {
      tour.push_back(0);  // truck is full: go to depot
      filled = 0;
    }
    tour.push_back(v);
    filled += demands[v];
    assert(filled <= capacity);
  }
  tour.push_back(0);  // go back to the depot

  float fitness = 0;
  for (unsigned i = 1; i < tour.size(); ++i)
    fitness += distances[tour[i - 1] * (clen + 1) + tour[i]];

  return Solution(*this, fitness, tour);
}

void CvrpInstance::evaluateChromosomesOnHost(unsigned int numberOfChromosomes,
                                             const float* chromosomes,
                                             float* results) const {
  for (unsigned i = 0; i < numberOfChromosomes; ++i) {
    const float* chromosome = chromosomes + i * chromosomeLength();
    results[i] = convertChromosomeToSolution(chromosome).fitness;
  }
}

__global__ void initAlleleIndices(const float* chromosomes,
                                  const unsigned numberOfChromosomes,
                                  const unsigned chromosomeLength,
                                  CvrpInstance::Gene* dest,
                                  unsigned* indices) {
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= numberOfChromosomes)
    return;

  const auto offset = tid * chromosomeLength;
  for (unsigned i = 0; i < chromosomeLength; ++i) {
    dest[offset + i].value = chromosomes[offset + i];
    dest[offset + i].chromosomeIndex = tid;
    indices[offset + i] = i;
  }
}

__global__ void checkGenesSortedCorrectly(const unsigned numberOfChromosomes,
                                          const unsigned chromosomeLength,
                                          const float* chromosomes,
                                          const CvrpInstance::Gene* genes,
                                          const unsigned* indices) {
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= numberOfChromosomes)
    return;

  bool* seen = (bool*)malloc(chromosomeLength * sizeof(bool));
  for (int i = 0; i < chromosomeLength; ++i)
    seen[i] = false;

  const auto offset = tid * chromosomeLength;
  for (unsigned i = 0; i < chromosomeLength; ++i) {
    assert(i == 0 || genes[offset + i - 1].value <= genes[offset + i].value);
    assert(genes[offset + i].chromosomeIndex == tid);
    assert(genes[offset + i].value == chromosomes[offset + indices[offset + i]]);
    assert(indices[offset + i] < chromosomeLength);
    assert(!seen[indices[offset + i]]);
    seen[indices[offset + i]] = true;
  }

  free(seen);
}

__global__ void cvrpEvaluateChromosomesOnDevice(const unsigned* allIndices,
                                                const unsigned numberOfChromosomes,
                                                const unsigned chromosomeLength,
                                                const unsigned capacity,
                                                const float* distances,
                                                const unsigned* demands,
                                                float* results) {
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= numberOfChromosomes)
    return;

  const auto* indices = allIndices + tid * chromosomeLength;

  unsigned u = 0;  // start in the depot
  float fitness = 0;
  unsigned filled = 0;
  for (unsigned i = 0; i < chromosomeLength; ++i) {
    unsigned v = indices[i] + 1;
    if (filled + demands[v] > capacity) {
      fitness += distances[u];  // go back to the depot
      u = 0;
      filled = 0;
    }

    fitness += distances[u * (chromosomeLength + 1) + v];
    filled += demands[v];
    u = v;
    assert(filled <= capacity);
  }

  fitness += distances[u];  // go back to the depot
  results[tid] = fitness;
}

void CvrpInstance::evaluateChromosomesOnDevice(hipStream_t stream,
                                               unsigned numberOfChromosomes,
                                               const float* dChromosomes,
                                               float* dResults) const {
  const unsigned block = THREADS_PER_BLOCK;
  const unsigned grid = ceilDiv(numberOfChromosomes, block);
  const unsigned totalGenes = numberOfChromosomes * chromosomeLength();

  CvrpInstance::Gene* dGenes = nullptr;
  unsigned* dIndices = nullptr;
  CUDA_CHECK(hipMalloc(&dGenes, totalGenes * sizeof(CvrpInstance::Gene)));
  CUDA_CHECK(hipMalloc(&dIndices, totalGenes * sizeof(unsigned)));

  initAlleleIndices<<<grid, block, 0, stream>>>(dChromosomes, numberOfChromosomes, chromosomeLength(), dGenes,
                                                dIndices);

  thrust::device_ptr<CvrpInstance::Gene> genesPtr(dGenes);
  thrust::device_ptr<unsigned> indicesPtr(dIndices);
  thrust::stable_sort_by_key(thrust::cuda::par.on(stream), genesPtr, genesPtr + totalGenes, indicesPtr);
  checkGenesSortedCorrectly<<<grid, block, 0, stream>>>(numberOfChromosomes, chromosomeLength(), dChromosomes, dGenes,
                                                        dIndices);

  cvrpEvaluateChromosomesOnDevice<<<grid, block, 0, stream>>>(dIndices, numberOfChromosomes, chromosomeLength(),
                                                              capacity, dDistances, dDemands, dResults);

  CUDA_CHECK(hipFree(dGenes));
  CUDA_CHECK(hipFree(dIndices));
}

__global__ void cvrpEvaluateIndicesOnDevice(const ChromosomeGeneIdxPair* allIndices,
                                            const unsigned numberOfChromosomes,
                                            const unsigned chromosomeLength,
                                            const unsigned capacity,
                                            const float* distances,
                                            const unsigned* demands,
                                            float* results) {
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= numberOfChromosomes)
    return;

  const auto* indices = allIndices + tid * chromosomeLength;

#ifndef NDEBUG
  bool* seen = (bool*)malloc(chromosomeLength * sizeof(bool));
  for (int i = 0; i < chromosomeLength; ++i)
    seen[i] = false;
  for (int i = 0; i < chromosomeLength; ++i) {
    assert(tid == indices[i].chromosomeIdx);
    assert(indices[i].geneIdx < chromosomeLength);
    assert(!seen[indices[i].geneIdx]);
    seen[indices[i].geneIdx] = true;
  }
  free(seen);
#endif  // NDEBUG

  unsigned u = 0;  // start in the depot
  float fitness = 0;
  unsigned filled = 0;
  for (unsigned i = 0; i < chromosomeLength; ++i) {
    unsigned v = indices[i].geneIdx + 1;
    if (filled + demands[v] > capacity) {
      fitness += distances[u];  // go back to the depot
      u = 0;
      filled = 0;
    }

    fitness += distances[u * (chromosomeLength + 1) + v];
    filled += demands[v];
    u = v;
    assert(filled <= capacity);
  }

  fitness += distances[u];  // go back to the depot
  results[tid] = fitness;
}

void CvrpInstance::evaluateIndicesOnDevice(hipStream_t stream,
                                           unsigned numberOfChromosomes,
                                           const ChromosomeGeneIdxPair* dIndices,
                                           float* dResults) const {
  const unsigned block = THREADS_PER_BLOCK;
  const unsigned grid = ceilDiv(numberOfChromosomes, block);
  cvrpEvaluateIndicesOnDevice<<<grid, block, 0, stream>>>(dIndices, numberOfChromosomes, chromosomeLength(), capacity,
                                                          dDistances, dDemands, dResults);
}
