#include "hip/hip_runtime.h"
#include "CudaError.cuh"
#include "CudaUtils.hpp"
#include "Logger.hpp"

#include <hip/hip_runtime.h>

#include <cctype>

__global__ void deviceIota(unsigned* arr, unsigned n) {
  for (unsigned i = threadIdx.x; i < n; i += blockDim.x) arr[i] = i;
}

void cuda::iota(hipStream_t stream, unsigned* arr, unsigned n) {
  constexpr auto threads = 256;
  logger::debug("iota on", n, "elements to array", arr, "on stream", stream,
                "using", threads, "threads");
  deviceIota<<<1, threads, 0, stream>>>(arr, n);
  CUDA_CHECK_LAST();
}

__global__ void deviceIotaMod(unsigned* arr, unsigned n, unsigned k) {
  for (unsigned i = threadIdx.x; i < n; i += blockDim.x) arr[i] = i % k;
}

void cuda::iotaMod(hipStream_t stream, unsigned* arr, unsigned n, unsigned k) {
  constexpr auto threads = 256;
  logger::debug("iotaMod on", n, "elements mod", k, "to array", arr,
                "on stream", stream, "using", threads, "threads");
  deviceIotaMod<<<1, threads, 0, stream>>>(arr, n, k);
  CUDA_CHECK_LAST();
}

// Defined by the bb_segsort implementation.
template <class Key, class Value>
void bbSegSort(Key*, Value*, std::size_t, std::size_t);

void cuda::segSort(float* dKeys,
                   unsigned* dValues,
                   std::size_t size,
                   std::size_t step) {
  bbSegSort(dKeys, dValues, size, step);
  CUDA_CHECK_LAST();
}
