#include "hip/hip_runtime.h"
#include "aaa.cuh"

#define SIMPLE 1
#define TEMPLATE 2
#define TYPE SIMPLE

const uint TILE_DIM = 32;
const uint BLOCK_ROWS = 8;

template <class T>
__global__ void transposeKernel(T* dst, const T* src, uint n, uint m) {
  __shared__ T tile[TILE_DIM][TILE_DIM + 1];
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;

  for (uint i = 0; i < TILE_DIM; i += BLOCK_ROWS)
    if ((y + i) < n && x < m)
      tile[threadIdx.y + i][threadIdx.x] = src[(y + i) * m + x];
  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;
  y = blockIdx.x * TILE_DIM + threadIdx.y;
  for (uint i = 0; i < TILE_DIM; i += BLOCK_ROWS)
    if (x < n && (y + i) < m)
      dst[(y + i) * n + x] = tile[threadIdx.x][threadIdx.y + i];
}

template <class T>
void transpose(T* dst, T* src, uint n, uint m) {
  const dim3 grid(ceilDiv(m, TILE_DIM), ceilDiv(n, TILE_DIM));
  const dim3 block(TILE_DIM, BLOCK_ROWS);
  transposeKernel<<<grid, block>>>(dst, src, n, m);
}

#if TYPE == SIMPLE
struct Permutation {
  __host__ __device__ Permutation(uint* _p, uint _ncols, uint _k)
      : p(_p), ncols(_ncols), k(_k) {}

  virtual __host__ __device__ inline uint operator[](uint i) {
    return this->p[this->k * this->ncols + i];
  }

  uint* p;
  uint ncols;
  uint k;
};

struct PermutationT : public Permutation {
  __host__ __device__ PermutationT(uint* _p, uint _ncols, uint _k)
      : Permutation(_p, _ncols, _k) {}

  __host__ __device__ inline uint operator[](uint i) override {
    return this->p[i * this->ncols + this->k];
  }
};

struct PermutationU : public Permutation {
  __host__ __device__ PermutationU(uint* _p, uint _ncols, uint _k)
      : Permutation(_p, _ncols, _k) {}

  __host__ __device__ inline uint operator[](uint i) override {
    return this->p[i * this->ncols + (this->k - 1)];
  }
};

// struct IprPermutation : public Permutation {
//   __host__ __device__
//   IprPermutation(uint* _p, uint _ncols, uint _k, uint _g, uint _gl, uint _gr)
//       : Permutation(_p, _ncols, _k), g(_h), gl(_hl), gr(_hr) {}

//   __host__ __device__ inline uint operator[](uint i) override {
//     const auto id = gl <= i && i < gr ? g : k;
//     return this->p[i * this->ncols + id];
//   }

//   uint g;
//   uint gl;
//   uint gr;
// };
#elif TYPE == TEMPLATE
template <class T>
struct Permutation {
  __host__ __device__ Permutation(T* _p, uint _ncols, uint _k)
      : p(_p), ncols(_ncols), k(_k) {}

  virtual __host__ __device__ inline T operator[](uint i) {
    return this->p[this->k * this->ncols + i];
  }

  T* p;
  uint ncols;
  uint k;
};

template <class T>
struct PermutationT : public Permutation<T> {
  __host__ __device__ PermutationT(T* _p, uint _ncols, uint _k)
      : Permutation<T>(_p, _ncols, _k) {}

  __host__ __device__ inline T operator[](uint i) override {
    return this->p[i * this->ncols + this->k];
  }
};

template <class T>
struct PermutationU : public Permutation<T> {
  __host__ __device__ PermutationU(T* _p, uint _ncols, uint _k)
      : Permutation<T>(_p, _ncols, _k) {}

  __host__ __device__ inline T operator[](uint i) override {
    return this->p[i * this->ncols + (this->k - 1)];
  }
};
#else
#error Invalid TYPE
#endif

__global__ void decode(float* dResults,
                       const uint* dPermutation,
                       const uint n,
                       const uint len,
                       const float* dDistances) {
  const auto k = blockIdx.x * blockDim.x + threadIdx.x;
  if (k >= n) return;

  auto u = dPermutation[k * len];
  auto v = dPermutation[(k + 1) * len - 1];
  auto fitness = dDistances[u * len + v];
  for (uint i = 1; i < len; ++i) {
    u = dPermutation[k * len + i - 1];
    v = dPermutation[k * len + i];
    fitness += dDistances[u * len + v];
  }
  dResults[k] = fitness;
}

__global__ void decodeT(float* dResults,
                        const uint* dPermutation,
                        const uint n,
                        const uint len,
                        const float* dDistances) {
  const auto k = blockIdx.x * blockDim.x + threadIdx.x;
  if (k >= n) return;

  auto u = dPermutation[k];
  auto v = dPermutation[(len - 1) * n + k];
  auto fitness = dDistances[u * len + v];
  for (uint i = 1; i < len; ++i) {
    u = dPermutation[(i - 1) * n + k];
    v = dPermutation[i * n + k];
    fitness += dDistances[u * len + v];
  }
  dResults[k] = fitness;
}

__global__ void decodeAccessWrapper(float* dResults,
                                    uint* dPermutation,
                                    const uint n,
                                    const uint len,
                                    const float* dDistances) {
  const auto k = blockIdx.x * blockDim.x + threadIdx.x;
  if (k >= n) return;

#if TYPE == SIMPLE
  Permutation p(dPermutation, len, k);
#elif TYPE == TEMPLATE
  Permutation<uint> p(dPermutation, len, k);
#else
#error Invalid TYPE
#endif

  auto u = p[0];
  auto v = p[len - 1];
  auto fitness = dDistances[u * len + v];
  for (uint i = 1; i < len; ++i) {
    u = p[i - 1];
    v = p[i];
    fitness += dDistances[u * len + v];
  }
  dResults[k] = fitness;
}

__global__ void decodeAccessWrapperT(float* dResults,
                                     uint* dPermutation,
                                     const uint n,
                                     const uint len,
                                     const float* dDistances) {
  const auto k = blockIdx.x * blockDim.x + threadIdx.x;
  if (k >= n) return;

#if TYPE == SIMPLE
  PermutationT p(dPermutation, n, k);
#elif TYPE == TEMPLATE
  PermutationT<uint> p(dPermutation, n, k);
#else
#error Invalid TYPE
#endif

  auto u = p[0];
  auto v = p[len - 1];
  auto fitness = dDistances[u * len + v];
  for (uint i = 1; i < len; ++i) {
    u = p[i - 1];
    v = p[i];
    fitness += dDistances[u * len + v];
  }
  dResults[k] = fitness;
}

__global__ void decodeAccessWrapperU(float* dResults,
                                     uint* dPermutation,
                                     const uint n,
                                     const uint len,
                                     const float* dDistances) {
  const auto k = blockIdx.x * blockDim.x + threadIdx.x;
  if (k >= n) return;

#if TYPE == SIMPLE
  PermutationU p(dPermutation, n, k + 1);
#elif TYPE == TEMPLATE
  PermutationU<uint> p(dPermutation, n, k + 1);
#else
#error Invalid TYPE
#endif

  auto u = p[0];
  auto v = p[len - 1];
  auto fitness = dDistances[u * len + v];
  for (uint i = 1; i < len; ++i) {
    u = p[i - 1];
    v = p[i];
    fitness += dDistances[u * len + v];
  }
  dResults[k] = fitness;
}

int main() {
  cout << "Running" << endl;
  cout << fixed << setprecision(9);
  cerr << fixed << setprecision(9);

  const uint n = 256;
  const uint len = 20000;
  const uint testCount = 30;
  alive;

  vector<uint> p(n * len);
  for (uint i = 0; i < n; ++i) {
    iota(p.begin() + i * len, p.begin() + (i + 1) * len, 0);
    random_shuffle(p.begin() + i * len, p.begin() + (i + 1) * len);
  }
  alive;

  uint* dp = nullptr;
  check(hipMalloc(&dp, n * len * sizeof(uint)));
  check(
      hipMemcpy(dp, p.data(), n * len * sizeof(uint), hipMemcpyHostToDevice));
  alive;

  uint* dpT = nullptr;
  check(hipMalloc(&dpT, n * len * sizeof(uint)));
  alive;

  mt19937 gen;
  uniform_real_distribution<float> random(1e2, 1e3);
  vector<float> distances(len * len);
  alive;
  for (uint i = 0; i < len; ++i) {
    for (uint j = i; j < len; ++j) {
      distances[i * len + j] = distances[j * len + i] = (i != j) * random(gen);
    }
  }
  alive;

  float* dDistances = nullptr;
  check(hipMalloc(&dDistances, len * len * sizeof(float)));
  check(hipMemcpy(dDistances, distances.data(), len * len * sizeof(float),
                   hipMemcpyHostToDevice));
  alive;

  float* dResults = nullptr;
  check(hipMalloc(&dResults, n * sizeof(float)));
  alive;

  hipEvent_t evBegin = nullptr;
  hipEvent_t evEnd = nullptr;
  float ms = -1;
  check(hipEventCreate(&evBegin));
  check(hipEventCreate(&evEnd));
  alive;

  decode<<<1, n>>>(dResults, dp, n, len, dDistances);
  check(last());
  ms = -1;
  check(hipEventRecord(evBegin, 0));
  for (uint i = 0; i < testCount; ++i) {
    decode<<<1, n>>>(dResults, dp, n, len, dDistances);
  }
  check(hipEventRecord(evEnd, 0));
  check(hipEventSynchronize(evEnd));
  check(hipEventElapsedTime(&ms, evBegin, evEnd));
  cout << "Decode: " << ms << "ms" << endl;

  vector<float> results(n);
  check(hipMemcpy(results.data(), dResults, n * sizeof(float),
                   hipMemcpyDeviceToHost));
  alive;

  // reset the values of dResults
  vector<float> resultsT(n, -1);
  check(hipMemcpy(dResults, resultsT.data(), n * sizeof(float),
                   hipMemcpyHostToDevice));
  alive;

  transpose(dpT, dp, n, len);
  check(last());
  decodeT<<<1, n>>>(dResults, dpT, n, len, dDistances);
  check(last());
  alive;

  ms = -1;
  check(hipEventRecord(evBegin, 0));
  for (uint i = 0; i < testCount; ++i) {
    transpose(dpT, dp, n, len);
    decodeT<<<1, n>>>(dResults, dpT, n, len, dDistances);
  }
  check(hipEventRecord(evEnd, 0));
  check(hipEventSynchronize(evEnd));
  check(hipEventElapsedTime(&ms, evBegin, evEnd));
  cout << "Transposed: " << ms << "ms" << endl;
  alive;

  check(hipMemcpy(resultsT.data(), dResults, n * sizeof(float),
                   hipMemcpyDeviceToHost));
  alive;

  for (uint i = 0; i < n; ++i) {
    if (cmp(results[i], resultsT[i]) != 0) {
      cerr << "Error: " << results[i] << " != " << resultsT[i] << endl;
    }
  }
  alive;

  // reset the values of dResults
  vector<float> resultsAW(n, -1);
  check(hipMemcpy(dResults, resultsAW.data(), n * sizeof(float),
                   hipMemcpyHostToDevice));
  alive;

  decodeAccessWrapper<<<1, n>>>(dResults, dp, n, len, dDistances);
  check(last());
  ms = -1;
  check(hipEventRecord(evBegin, 0));
  for (uint i = 0; i < testCount; ++i) {
    decodeAccessWrapper<<<1, n>>>(dResults, dp, n, len, dDistances);
  }
  check(hipEventRecord(evEnd, 0));
  check(hipEventSynchronize(evEnd));
  check(hipEventElapsedTime(&ms, evBegin, evEnd));
  cout << "Decode AW: " << ms << "ms" << endl;

  check(hipMemcpy(resultsAW.data(), dResults, n * sizeof(float),
                   hipMemcpyDeviceToHost));
  alive;

  for (uint i = 0; i < n; ++i) {
    if (cmp(results[i], resultsAW[i]) != 0) {
      cerr << "Error: " << results[i] << " != " << resultsAW[i] << endl;
    }
    // cout << results[i] << '\n';
  }

  // reset the values of dResults
  vector<float> resultsAWT(n, -1);
  check(hipMemcpy(dResults, resultsAWT.data(), n * sizeof(float),
                   hipMemcpyHostToDevice));
  alive;

  transpose(dpT, dp, n, len);
  check(last());
  decodeAccessWrapperT<<<1, n>>>(dResults, dpT, n, len, dDistances);
  check(last());
  alive;

  ms = -1;
  check(hipEventRecord(evBegin, 0));
  for (uint i = 0; i < testCount; ++i) {
    transpose(dpT, dp, n, len);
    decodeAccessWrapperT<<<1, n>>>(dResults, dpT, n, len, dDistances);
  }
  check(hipEventRecord(evEnd, 0));
  check(hipEventSynchronize(evEnd));
  check(hipEventElapsedTime(&ms, evBegin, evEnd));
  cout << "Transposed AW: " << ms << "ms" << endl;
  alive;

  check(hipMemcpy(resultsAWT.data(), dResults, n * sizeof(float),
                   hipMemcpyDeviceToHost));
  alive;

  for (uint i = 0; i < n; ++i) {
    if (cmp(results[i], resultsAWT[i]) != 0) {
      cerr << "Error: " << results[i] << " != " << resultsAWT[i] << endl;
    }
    // cout << results[i] << '\n';
  }

  // Test if it works with many implementations
  decodeAccessWrapperU<<<1, n>>>(dResults, dpT, n, len, dDistances);

  cout << "Done!" << endl;

  check(hipFree(dp));
  check(hipFree(dpT));
  check(hipFree(dDistances));
  check(hipFree(dResults));
  check(hipEventDestroy(evBegin));
  check(hipEventDestroy(evEnd));

  return 0;
}
