#include "hip/hip_runtime.h"
#include "CudaError.cuh"
#include "CudaUtils.hpp"
#include "Logger.hpp"

#include <hip/hip_runtime.h>

#include <cctype>

__global__ void deviceIota(unsigned* arr, unsigned n) {
  for (unsigned i = threadIdx.x; i < n; i += blockDim.x) arr[i] = i;
}

void box::cuda::iota(hipStream_t stream, unsigned* arr, unsigned n) {
  constexpr auto threads = 256;
  box::logger::debug("iota on", n, "elements to array", arr, "on stream", stream,
                "using", threads, "threads");
  deviceIota<<<1, threads, 0, stream>>>(arr, n);
  CUDA_CHECK_LAST();
}

__global__ void deviceIotaMod(unsigned* arr, unsigned n, unsigned k) {
  for (unsigned i = threadIdx.x; i < n; i += blockDim.x) arr[i] = i % k;
}

void box::cuda::iotaMod(hipStream_t stream,
                        unsigned* arr,
                        unsigned n,
                        unsigned k) {
  constexpr auto threads = 256;
  box::logger::debug("iotaMod on", n, "elements mod", k, "to array", arr,
                "on stream", stream, "using", threads, "threads");
  deviceIotaMod<<<1, threads, 0, stream>>>(arr, n, k);
  CUDA_CHECK_LAST();
}

// Defined by the bb_segsort implementation.
template <class Key, class Value>
void bbSegSort(Key*, Value*, std::size_t, std::size_t);

void box::cuda::segSort(float* dKeys,
                        unsigned* dValues,
                        std::size_t segCount,
                        std::size_t segSize) {
  bbSegSort(dKeys, dValues, segCount, segSize);
  CUDA_CHECK_LAST();
}
